
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <cfloat>
#include <cmath>
#include <iostream>
#include <vector>
#define BLOCK_SIZE 16
#define TILE_WIDTH 16

#define FLOAT_LOWEST -340282346638528859811704183484516925440.0

__global__ void similarityMatrixAndTranspose(
    const float* descriptors0, const float* descriptors1, int nDescriptors0,
    const int nDescriptors1, const int descriptorDim, float* sim, float* simT) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;

  if (idx < nDescriptors0 && idy < nDescriptors1) {
    float dotProduct = 0.0f;
    for (int i = 0; i < descriptorDim; ++i) {
      dotProduct += descriptors0[idx * descriptorDim + i] *
                    descriptors1[idy * descriptorDim + i];
    }

    sim[idx * nDescriptors1 + idy] = dotProduct;
    simT[idy * nDescriptors0 + idx] = dotProduct;
  }
}

__global__ void matrixMultiplyShared(const float* A, const float* B, float* C,
                                     float* CT, int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  __shared__ float sA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float sB[TILE_WIDTH][TILE_WIDTH];

  int Row = blockDim.y * blockIdx.y + threadIdx.y;
  int Col = blockDim.x * blockIdx.x + threadIdx.x;
  float Cvalue = 0.0;
  sA[threadIdx.y][threadIdx.x] = 0.0;
  sB[threadIdx.y][threadIdx.x] = 0.0;

  for (int ph = 0; ph < (((numAColumns - 1) / TILE_WIDTH) + 1); ph++) {
    if ((Row < numARows) && (threadIdx.x + (ph * TILE_WIDTH)) < numAColumns) {
      sA[threadIdx.y][threadIdx.x] =
          A[(Row * numAColumns) + threadIdx.x + (ph * TILE_WIDTH)];
    } else {
      sA[threadIdx.y][threadIdx.x] = 0.0;
    }
    if (Col < numBColumns && (threadIdx.y + ph * TILE_WIDTH) < numBRows) {
      sB[threadIdx.y][threadIdx.x] =
          B[(threadIdx.y + ph * TILE_WIDTH) * numBColumns + Col];
    } else {
      sB[threadIdx.y][threadIdx.x] = 0.0;
    }
    __syncthreads();

    for (int j = 0; j < TILE_WIDTH; ++j) {
      Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
    }
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Cvalue;
    CT[Col * numCRows + Row] = Cvalue;
  }
}

void simCublas(const float* descriptors0, const float* descriptors1,
               int nDescriptors0, int nDescriptors1, int descriptorDim,
               float* sim) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasStatus_t status =
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nDescriptors1,
                  nDescriptors0, 128, &alpha, descriptors1, nDescriptors1,
                  descriptors0, nDescriptors0, &beta, sim, nDescriptors1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cout << "hipblasSgemm failed" << std::endl;
  }

  hipblasDestroy(handle);
}

__global__ void similarityMatrixAndTransposeV2(
    const float* descriptors0, const float* descriptors1, int nDescriptors0,
    int nDescriptors1, int descriptorDim, float* sim, float* simT) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;

  if (idx < nDescriptors0 && idy < nDescriptors1) {
    int globalIdx = idx * descriptorDim;
    int globalIdy = idy * descriptorDim;

    float dotProduct = 0.0f;
#pragma unroll 4
    for (int i = 0; i < descriptorDim; i += 4) {
      float4 vec0 = *((float4*)(descriptors0 + globalIdx + i));
      float4 vec1 = *((float4*)(descriptors1 + globalIdy + i));

      dotProduct += __fmaf_rn(vec0.x, vec1.x, 0.0f);
      dotProduct += __fmaf_rn(vec0.y, vec1.y, 0.0f);
      dotProduct += __fmaf_rn(vec0.z, vec1.z, 0.0f);
      dotProduct += __fmaf_rn(vec0.w, vec1.w, 0.0f);
    }

    sim[idx * nDescriptors1 + idy] = dotProduct;
    simT[idy * nDescriptors0 + idx] = dotProduct;
  }
}

__global__ void similarityMatrixFast(const float* descriptors0,
                                     const float* descriptors1,
                                     int nDescriptors0, int nDescriptors1,
                                     int descriptorDim, float* sim) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;

  if (idx < nDescriptors0 && idy < nDescriptors1) {
    int globalIdx = idx * descriptorDim;
    int globalIdy = idy * descriptorDim;

    float dotProduct = 0.0f;
#pragma unroll 4
    for (int i = 0; i < descriptorDim; i += 4) {
      float4 vec0 = *((float4*)(descriptors0 + globalIdx + i));
      float4 vec1 = *((float4*)(descriptors1 + globalIdy + i));

      dotProduct += __fmaf_rn(vec0.x, vec1.x, 0.0f);
      dotProduct += __fmaf_rn(vec0.y, vec1.y, 0.0f);
      dotProduct += __fmaf_rn(vec0.z, vec1.z, 0.0f);
      dotProduct += __fmaf_rn(vec0.w, vec1.w, 0.0f);
    }

    sim[idx * nDescriptors1 + idy] = dotProduct;
  }
}

__global__ void transposeSim(const float* sim, float* simT, int nDescriptors0,
                             int nDescriptors1) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;

  if (idx < nDescriptors0 && idy < nDescriptors1) {
    simT[idy * nDescriptors0 + idx] = sim[idx * nDescriptors1 + idy];
  }
}

__global__ void transposeDescriptors(const float* descriptors,
                                     float* descriptorsT, int nDescriptors,
                                     int descriptorDim) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;

  if (idx < nDescriptors && idy < descriptorDim) {
    descriptorsT[idy * nDescriptors + idx] =
        descriptors[idx * descriptorDim + idy];
  }
}

__global__ void find_nn(const float* sim, int* matches, float* scores,
                        const int nDescriptors0, const int nDescriptors1,
                        const float ratio_thresh_sq) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors0) {
    float sim_nn0 = -1e30f;
    float sim_nn1 = -1e30f;
    int nearestNeighborIdx = -1;
#pragma unroll 2
    for (int i = 0; i < nDescriptors1; ++i) {
      if (sim[idx * nDescriptors1 + i] > sim_nn0) {
        sim_nn1 = sim_nn0;
        sim_nn0 = sim[idx * nDescriptors1 + i];
        nearestNeighborIdx = i;
      } else if (sim[idx * nDescriptors1 + i] > sim_nn1) {
        sim_nn1 = sim[idx * nDescriptors1 + i];
      }
    }

    float dist_nn0 = 2 * (1 - sim_nn0);
    float dist_nn1 = 2 * (1 - sim_nn1);

    bool validMatch = (dist_nn0 <= ratio_thresh_sq * dist_nn1);

    matches[idx] = (validMatch) ? nearestNeighborIdx : -1;
    scores[idx] = (validMatch) ? (sim_nn0 + 1) / 2.0f : 0.0f;
  }
}

__global__ void find_nnV2(const float* sim, int* matches, float* scores,
                          const int nDescriptors0, const int nDescriptors1,
                          const float ratio_thresh_sq) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors0) {
    float sim_nn0 = -1e30f;
    float sim_nn1 = -1e30f;
    int nearestNeighborIdx = -1;
    for (int i = 0; i < nDescriptors1; ++i) {
      float current_sim = sim[idx * nDescriptors1 + i];

      if (current_sim > sim_nn0) {
        sim_nn1 = sim_nn0;
        sim_nn0 = current_sim;
        nearestNeighborIdx = i;
      } else if (current_sim > sim_nn1) {
        sim_nn1 = current_sim;
      }
    }

    float dist_nn0 = 2 * (1 - sim_nn0);
    float dist_nn1 = 2 * (1 - sim_nn1);

    bool validMatch = (dist_nn0 <= ratio_thresh_sq * dist_nn1);

    matches[idx] = (validMatch) ? nearestNeighborIdx : -1;
    scores[idx] = (validMatch) ? (sim_nn0 + 1) / 2.0f : 0.0f;
  }
}

__global__ void findNearestNeighbors(const float* descriptors0,
                                     const float* descriptors1, int* matches,
                                     float* scores, int nDescriptors0,
                                     int nDescriptors1,
                                     const float ratio_thresh_sq,
                                     float distance_thresh_sq) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors0) {
    float sim0 = -100000.0f;
    int nearestNeighborIdx = -1;

    for (int i = 0; i < nDescriptors1; ++i) {
      float distance = 0.0f;
      for (int j = 0; j < 128; ++j) {
        float diff = descriptors0[idx * 128 + j] * descriptors1[i * 128 + j];
        distance += diff * diff;
      }
      if (distance > sim0) {
        sim0 = distance;
        nearestNeighborIdx = i;
      }
    }

    float sim1 = -100000.0f;
    int nearestNeighborIdx1 = -1;

    for (int i = 0; i < nDescriptors1; ++i) {
      float distance = 0.0f;
      for (int j = 0; j < 128; ++j) {
        if (i == nearestNeighborIdx) {
          continue;
        }
        float diff = descriptors0[idx * 128 + j] * descriptors1[i * 128 + j];
        distance += diff;
      }
      if (distance > sim1) {
        sim1 = distance;
        nearestNeighborIdx1 = i;
      }
    }

    // float dist_nn0 = (2 * (1 - sim0));
    // float dist_nn1 = 2 * (1 - sim1);

    bool validMatch = ((2 * (1 - sim0)) <= ratio_thresh_sq * (2 * (1 - sim1)));

    matches[idx] = (validMatch) ? nearestNeighborIdx : -1;
    scores[idx] = (validMatch) ? (sim0 + 1) / 2.0f : 0.0f;
  }
}
__global__ void mutualCheck(const int* matches0, const int* matches1,
                            int* matches, int nDescriptors) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors) {
    int match1 = matches0[idx];
    int match2 = (match1 != -1) ? matches1[match1] : -1;
    matches[idx] = (match2 == idx) ? match1 : -1;
  }
}

__global__ void mutualCheckV2(int* matches0, const int* matches1,
                              int nDescriptors) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors) {
    int match1 = matches0[idx];
    int match2 = (match1 != -1) ? matches1[match1] : -1;
    matches0[idx] = (match2 == idx) ? match1 : -1;
  }
}

__global__ void mutualCheckV3(int* matches0, const int* matches1,
                              int nDescriptors) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < nDescriptors) {
    int match1 = matches0[idx];
    matches0[idx] = (match1 != -1 && matches1[match1] == idx) ? match1 : -1;
  }
}

void allocateDescriptors(float** d_descriptors,
                         const std::vector<float>& descriptors) {
  hipMalloc(d_descriptors, descriptors.size() * sizeof(float));
  hipMemcpy(*d_descriptors, descriptors.data(),
             descriptors.size() * sizeof(float), hipMemcpyHostToDevice);
}

void featureMatchingLegacy(const float* d_descriptors0,
                           const float* d_descriptors1,
                           std::vector<int>& matches,
                           std::vector<float>& scores, float ratio_thresh_sq,
                           int nDescriptors0, int nDescriptors1) {
  float* d_sim;
  float* d_simT;

  hipMallocAsync(&d_sim, nDescriptors0 * nDescriptors1 * sizeof(float), 0);
  hipMallocAsync(&d_simT, nDescriptors0 * nDescriptors1 * sizeof(float), 0);

  int *d_matches0, *d_matches1;
  float *d_scores0, *d_scores1;

  int threadsPerBlock = 128;
  dim3 threadsPerBlock2D(8, 8);

  int blocksPerGrid = (nDescriptors0 + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridT = (nDescriptors1 + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blocksPerGrid2D(
      (nDescriptors0 + threadsPerBlock2D.x - 1) / threadsPerBlock2D.x,
      (nDescriptors1 + threadsPerBlock2D.y - 1) / threadsPerBlock2D.y);

  hipDeviceSynchronize();

  hipMallocAsync(&d_matches0, nDescriptors0 * sizeof(int), 0);
  hipMallocAsync(&d_matches1, nDescriptors1 * sizeof(int), 0);
  hipMallocAsync(&d_scores0, nDescriptors0 * sizeof(float), 0);
  hipMallocAsync(&d_scores1, nDescriptors1 * sizeof(float), 0);

  similarityMatrixAndTransposeV2<<<blocksPerGrid2D, threadsPerBlock2D>>>(
      d_descriptors0, d_descriptors1, nDescriptors0, nDescriptors1, 128, d_sim,
      d_simT);

  hipDeviceSynchronize();

  find_nnV2<<<blocksPerGrid, threadsPerBlock>>>(d_sim, d_matches0, d_scores0,
                                                nDescriptors0, nDescriptors1,
                                                ratio_thresh_sq);

  find_nnV2<<<blocksPerGridT, threadsPerBlock>>>(d_simT, d_matches1, d_scores1,
                                                 nDescriptors1, nDescriptors0,
                                                 ratio_thresh_sq);

  hipDeviceSynchronize();

  hipMemcpyAsync(scores.data(), d_scores0, nDescriptors0 * sizeof(float),
                  hipMemcpyDeviceToHost);

  hipFreeAsync(d_sim, 0);
  hipFreeAsync(d_simT, 0);
  hipFreeAsync(d_scores1, 0);

  mutualCheckV3<<<blocksPerGrid, threadsPerBlock>>>(d_matches0, d_matches1,
                                                    nDescriptors0);

  hipDeviceSynchronize();

  hipMemcpyAsync(matches.data(), d_matches0, nDescriptors0 * sizeof(int),
                  hipMemcpyDeviceToHost);

  hipFreeAsync(d_matches0, 0);
  hipFreeAsync(d_matches1, 0);
  hipFreeAsync(d_scores0, 0);

  hipDeviceSynchronize();
}

void featureMatching(const float* d_descriptors0, const float* d_descriptors1,
                     std::vector<int>& matches, std::vector<float>& scores,
                     float ratio_thresh_sq, int nDescriptors0,
                     int nDescriptors1) {
  float* d_sim;
  float* d_simT;

  hipMallocAsync(&d_sim, nDescriptors0 * nDescriptors1 * sizeof(float), 0);
  hipMallocAsync(&d_simT, nDescriptors0 * nDescriptors1 * sizeof(float), 0);

  int *d_matches0, *d_matches1;
  float *d_scores0, *d_scores1;

  hipDeviceSynchronize();

  hipMallocAsync(&d_matches0, nDescriptors0 * sizeof(int), 0);
  hipMallocAsync(&d_matches1, nDescriptors1 * sizeof(int), 0);
  hipMallocAsync(&d_scores0, nDescriptors0 * sizeof(float), 0);
  hipMallocAsync(&d_scores1, nDescriptors1 * sizeof(float), 0);

  float* d_descriptors1T;
  hipMalloc(&d_descriptors1T, nDescriptors1 * 128 * sizeof(float));

  hipDeviceSynchronize();

  dim3 threadsPerBlock2Ddt(8, 8);
  dim3 blocksPerGrid2Ddt(
      (nDescriptors1 + threadsPerBlock2Ddt.x - 1) / threadsPerBlock2Ddt.x,
      (128 + threadsPerBlock2Ddt.y - 1) / threadsPerBlock2Ddt.y);

  transposeDescriptors<<<blocksPerGrid2Ddt, threadsPerBlock2Ddt>>>(
      d_descriptors1, d_descriptors1T, nDescriptors1, 128);

  hipDeviceSynchronize();

  dim3 threadsPerBlock2Dmult(TILE_WIDTH, TILE_WIDTH);
  dim3 blocksPerGrid2Dmult((nDescriptors1 / TILE_WIDTH + 1),
                           (nDescriptors0 / TILE_WIDTH + 1));

  matrixMultiplyShared<<<blocksPerGrid2Dmult, threadsPerBlock2Dmult>>>(
      d_descriptors0, d_descriptors1T, d_sim, d_simT, nDescriptors0, 128, 128,
      nDescriptors1, nDescriptors0, nDescriptors1);

  hipDeviceSynchronize();

  int threadsPerBlock = TILE_WIDTH;
  int blocksPerGrid = (nDescriptors0 + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridT = (nDescriptors1 + threadsPerBlock - 1) / threadsPerBlock;

  find_nnV2<<<blocksPerGrid, threadsPerBlock>>>(d_sim, d_matches0, d_scores0,
                                                nDescriptors0, nDescriptors1,
                                                ratio_thresh_sq);

  find_nnV2<<<blocksPerGridT, threadsPerBlock>>>(d_simT, d_matches1, d_scores1,
                                                 nDescriptors1, nDescriptors0,
                                                 ratio_thresh_sq);

  hipDeviceSynchronize();

  hipMemcpyAsync(scores.data(), d_scores0, nDescriptors0 * sizeof(float),
                  hipMemcpyDeviceToHost);

  hipFreeAsync(d_descriptors1T, 0);
  hipFreeAsync(d_sim, 0);
  hipFreeAsync(d_simT, 0);
  hipFreeAsync(d_scores1, 0);

  mutualCheckV3<<<blocksPerGrid, threadsPerBlock>>>(d_matches0, d_matches1,
                                                    nDescriptors0);

  hipDeviceSynchronize();

  hipMemcpyAsync(matches.data(), d_matches0, nDescriptors0 * sizeof(int),
                  hipMemcpyDeviceToHost);

  hipFreeAsync(d_matches0, 0);
  hipFreeAsync(d_matches1, 0);
  hipFreeAsync(d_scores0, 0);

  hipDeviceSynchronize();
}